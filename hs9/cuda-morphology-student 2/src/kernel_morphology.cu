#include "hip/hip_runtime.h"
#include "kernel_morphology.h"
#include <hip/hip_runtime.h>

/* Original version
//Computes the minimum value in the neighborhood.
void kernel_erosion(const unsigned char* Frame_in, unsigned char* Frame_out, int height, int width, int* Mask, int mask_radius)
{
    int x = 0;
    int y = 0;

    for (y = 0; y < height; y++) {
        for (x = 0; x < width; x++) {

            // Dummy copy code
            Frame_out[y * width + x] = Frame_in[y * width + x];

        }
    }
}
//Computes the maximum value in the neighborhood.
void kernel_dilation(const unsigned char* Frame_in, unsigned char* Frame_out, int height, int width, int* Mask, int mask_radius)
{
    int x = 0;
    int y = 0;

    for (y = 0; y < height; y++) {
        for (x = 0; x < width; x++) {

            // Dummy copy code
            Frame_out[y * width + x] = Frame_in[y * width + x];

        }
    }
}
*/

/* Version 2:
 * Directly limit the neighborhood coordinates through min(max(...)) ,
 * without processing the boundary pixels separately outside the main loop.
 *
 * Each time the neighborhood is traversed, boundary checking and Mask judgment are performed, 
 * which increases the amount of calculation.
*/
/*
__global__ 
void kernel_erosion(const unsigned char* Frame_in, unsigned char* Frame_out, int height, int width, int* Mask, int mask_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int min_val = 255;
        for (int dy = -mask_radius; dy <= mask_radius; dy++) {
            for (int dx = -mask_radius; dx <= mask_radius; dx++) {
                if (Mask[(dy + mask_radius) * (2 * mask_radius + 1) + (dx + mask_radius)]) {
                    int nx = min(max(x + dx, 0), width - 1);
                    int ny = min(max(y + dy, 0), height - 1);
                    min_val = min(min_val, (int)Frame_in[ny * width + nx]);
                }
            }
        }
        Frame_out[y * width + x] = (unsigned char)min_val;
    }
}

__global__ 
void kernel_dilation(const unsigned char* Frame_in, unsigned char* Frame_out, int height, int width, int* Mask, int mask_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int max_val = 0;
        for (int dy = -mask_radius; dy <= mask_radius; dy++) {
            for (int dx = -mask_radius; dx <= mask_radius; dx++) {
                if (Mask[(dy + mask_radius) * (2 * mask_radius + 1) + (dx + mask_radius)]) {
                    int nx = min(max(x + dx, 0), width - 1);
                    int ny = min(max(y + dy, 0), height - 1);
                    max_val = max(max_val, (int)Frame_in[ny * width + nx]);
                }
            }
        }
        Frame_out[y * width + x] = (unsigned char)max_val;
    }
}
*/

/* Version 3:
 * Process boundaries separately for higher performance.
 */

__global__
void kernel_erosion(const unsigned char* Frame_in, unsigned char* Frame_out, int height, int width, int* Mask, int mask_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    if (x >= width || y >= height) return;
    // Processing boundary pixels: directly copy the input pixel value to the output
    if (x < mask_radius || x >= width - mask_radius || y < mask_radius || y >= height - mask_radius) {
        Frame_out[y * width + x] = Frame_in[y * width + x];
        return;
    }

    int min_val = 255;
    for (int dy = -mask_radius; dy <= mask_radius; dy++) {
        //If the value of Mask is 1, the corresponding neighboring pixels participate in the calculation
        for (int dx = -mask_radius; dx <= mask_radius; dx++) {
            if (Mask[(dy + mask_radius) * (2 * mask_radius + 1) + (dx + mask_radius)]) {
                int neighbor_val = Frame_in[(y + dy) * width + (x + dx)];
                min_val = min(min_val, neighbor_val);
            }
        }
    }
    Frame_out[y * width + x] = (unsigned char)min_val;
}

__global__
void kernel_dilation(const unsigned char* Frame_in, unsigned char* Frame_out, int height, int width, int* Mask, int mask_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    if (x >= width || y >= height) return;

    // Processing boundary pixels: directly copy the input pixel value to the output
    if (x < mask_radius || x >= width - mask_radius || y < mask_radius || y >= height - mask_radius) {
        Frame_out[y * width + x] = Frame_in[y * width + x];
        return;
    }
    int max_val = 0;

    for (int dy = -mask_radius; dy <= mask_radius; dy++) {
        for (int dx = -mask_radius; dx <= mask_radius; dx++) {
            if (Mask[(dy + mask_radius) * (2 * mask_radius + 1) + (dx + mask_radius)]) {
                int neighbor_val = Frame_in[(y + dy) * width + (x + dx)];
                max_val = max(max_val, neighbor_val);
            }
        }
    }

    Frame_out[y * width + x] = (unsigned char)max_val;
}

#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>

const float K = 1.0f;     
const float S0 = 1.0f;    // the spot values
const float v0 = 0.1f;  
const float r = 0.0f;     // the risk-free interest rate
const float kappa = 0.5f; // the mean reversion rate of the volatility
const float theta = 0.1f; // the long-term volatility
const float sigma = 0.3f; // the volatility of volatility
const float rho = -0.7f;  
const int T = 1;          
const int steps = 1000;   
const float dt = 1.0f / steps; 
const int simulations = 100000; 

// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


__global__ void hestonMonteCarlo(float *d_results, int steps, float dt, float kappa, float theta, float sigma, float rho) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //Initialize the random number generator
    hiprandState state;
    hiprand_init(1234, tid, 0, &state);

    float St = S0;
    float vt = v0;

    //Simulation time step
    for (int i = 0; i < steps; ++i) {
        float G1 = hiprand_normal2(&state);
        float G2 = hiprand_normal2(&state);

        // Calculate the delta of asset price and volatility
        float dSt = r * St * dt + sqrtf(vt) * St * sqrtf(dt) * (rho * G1 + sqrtf(1 - rho * rho) * G2);
        float dvt = kappa * (theta - vt) * dt + sigma * sqrtf(vt) * sqrtf(dt) * G1;

        St += dSt;
        vt = fabs(vt + dvt); // the function g is either taken to be equal to (·)+ or to | · |
    }
    // E[f(ST )] = E[(S1 − 1)+].
    d_results[tid] = fmaxf(St - K, 0.0f);
}


int main() {
    //Allocate memory on the device to store the results
    float *d_results;
    hipMalloc((void **)&d_results, simulations * sizeof(float));

    int threadsPerBlock = 256;
    int blocks = (simulations + threadsPerBlock - 1) / threadsPerBlock;

	float Tim;
	hipEvent_t start, stop;			// GPU timer instructions
	hipEventCreate(&start);			// GPU timer instructions
	hipEventCreate(&stop);				// GPU timer instructions
	hipEventRecord(start, 0);			// GPU timer instructions

    hestonMonteCarlo<<<blocks, threadsPerBlock>>>(d_results, steps, dt, kappa, theta, sigma, rho);

	hipEventRecord(stop, 0);			// GPU timer instructions
	hipEventSynchronize(stop);			// GPU timer instructions
	hipEventElapsedTime(&Tim,			// GPU timer instructions
		start, stop);					// GPU timer instructions
	hipEventDestroy(start);			// GPU timer instructions
	hipEventDestroy(stop);				// GPU timer instructions

    float *h_results = (float *)malloc(simulations * sizeof(float));
    hipMemcpy(h_results, d_results, simulations * sizeof(float), hipMemcpyDeviceToHost);

    float option_price = 0.0f;
    for (int i = 0; i < simulations; ++i) {
        option_price += h_results[i];
    }
    option_price /= simulations;
    option_price *= expf(-r * T); 

    std::cout << "Option Price: " << option_price << std::endl;

    free(h_results);
    hipFree(d_results);

    return 0;
}

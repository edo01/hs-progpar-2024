
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>

const float K = 1.0f;     
const float S0 = 1.0f;    // the spot values
const float v0 = 0.1f;  
const float r = 0.0f;     // the risk-free interest rate
const float kappa = 0.5f; // the mean reversion rate of the volatility
const float theta = 0.1f; // the long-term volatility
const float sigma = 0.3f; // the volatility of volatility
const float rho = -0.7f;  
const int T = 1;          
const int steps = 1000;   
const float dt = 1.0f / steps; 
const int simulations = 100000; 

// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

/*One-Dimensional Normal Law. Cumulative distribution function. */
double NP(double x) {
	const double p = 0.2316419;
	const double b1 = 0.319381530;
	const double b2 = -0.356563782;
	const double b3 = 1.781477937;
	const double b4 = -1.821255978;
	const double b5 = 1.330274429;
	const double one_over_twopi = 0.39894228;
	double t;

	if (x >= 0.0) {
		t = 1.0 / (1.0 + p * x);
		return (1.0 - one_over_twopi * exp(-x * x / 2.0) * t * (t * (t *
			(t * (t * b5 + b4) + b3) + b2) + b1));
	}
	else {/* x < 0 */
		t = 1.0 / (1.0 - p * x);
		return (one_over_twopi * exp(-x * x / 2.0) * t * (t * (t * (t *
			(t * b5 + b4) + b3) + b2) + b1));
	}
}

// Set the state for each thread
__global__ void init_curand_state(hiprandState_t* state)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	/* Each thread gets same seed, a different sequence
	   number, no offset */
	hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void hestonMonteCarlo(float *d_results, int steps, float dt, float kappa, float theta, float sigma, float rho, hiprandState_t* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //Initialize the random number generator
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState localState = state[idx];

    float St = S0;
    float vt = v0;

    //Simulation time step
    for (int i = 0; i < steps; ++i) {
        float G1 = hiprand_normal(&localState);
        float G2 = hiprand_normal(&localState);

        // Calculate the delta of asset price and volatility
        float dSt = r * St * dt + sqrtf(vt) * St * sqrtf(dt) * (rho * G1 + sqrtf(1 - rho * rho) * G2);
        float dvt = kappa * (theta - vt) * dt + sigma * sqrtf(vt) * sqrtf(dt) * G1;

        St += dSt;
        vt = fabs(vt + dvt); // the function g is either taken to be equal to (·)+ or to | · |
    }
    // E[f(ST )] = E[(S1 − 1)+].
    d_results[tid] = fmaxf(St - K, 0.0f);
}


int main() {
    
    int NTPB = 256;
    int NB = (simulations + NTPB - 1) / NTPB;
    
    //Allocate memory on the device to store the results
    float *d_results;
    hipMalloc((void **)&d_results, simulations * sizeof(float));

    hiprandState_t* state;
	// cudaMalloc the array state
	hipMalloc(&state, simulations * sizeof(hiprandState_t)); // is the total number of state

    init_curand_state<<<NB, NTPB>>>(state);

	float Tim;
	hipEvent_t start, stop;			// GPU timer instructions
	hipEventCreate(&start);			// GPU timer instructions
	hipEventCreate(&stop);				// GPU timer instructions
	hipEventRecord(start, 0);			// GPU timer instructions

    hestonMonteCarlo<<<NB, NTPB>>>(d_results, steps, dt, kappa, theta, sigma, rho, state);

	hipEventRecord(stop, 0);			// GPU timer instructions
	hipEventSynchronize(stop);			// GPU timer instructions
	hipEventElapsedTime(&Tim,			// GPU timer instructions
		start, stop);					// GPU timer instructions
	hipEventDestroy(start);			// GPU timer instructions
	hipEventDestroy(stop);				// GPU timer instructions

    float *h_results = (float *)malloc(simulations * sizeof(float));
    hipMemcpy(h_results, d_results, simulations * sizeof(float), hipMemcpyDeviceToHost);

    float option_price = 0.0f;
    for (int i = 0; i < simulations; ++i) {
        option_price += h_results[i];
    }
    option_price /= simulations;
    option_price *= expf(-r * T); 

    std::cout << "Option Price: " << option_price << std::endl;

    printf("The estimated price is equal to %f\n", option_price);

	printf("The true price %f\n", S0 * NP((r + 0.5 * sigma * sigma)/sigma) -
									K * expf(-r) * NP((r - 0.5 * sigma * sigma) / sigma));

	printf("Execution time %f ms\n", Tim);
    free(h_results);
    hipFree(d_results);

    return 0;
}

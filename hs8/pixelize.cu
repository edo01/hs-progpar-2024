#include "hip/hip_runtime.h"
#include "cppdefs.h"
#include "cuda_kernels.h"
EXTERN
{
#include "easypap.h"
}

EXTERN __global__ void pixelize_cuda_fake (uint32_t *img,
                                           unsigned DIM)
{
  unsigned index = gpu_get_index ();
  __shared__ uint32_t color;

  if (threadIdx.x + threadIdx.y == 0)
    color = img[index];

  __syncthreads ();

  img[index] = color;
}

__device__ static int4 int4_add (int4 a, int4 b)
{
  return make_int4 (a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__device__ static int4 int4_div_int (int4 a, int b)
{
  return make_int4 (a.x / b, a.y / b, a.z / b, a.w / b);
}

// We redefine the kernel launcher function because
// we use shared memory of variable length
EXTERN unsigned pixelize_compute_cuda_fake (unsigned nb_iter)
{
  hipError_t ret;
  dim3 grid  = {GPU_SIZE_X / TILE_W, GPU_SIZE_Y / TILE_H, 1};
  dim3 block = {TILE_W, TILE_H, 1};

  ret = hipSetDevice (cuda_device (0));
  check (ret, "hipSetDevice");

  uint64_t clock = monitoring_start_tile (easypap_gpu_lane (0));

  for (int i = 0; i < nb_iter; i++)
    pixelize_cuda<<<grid, block, 0,
                    hip_stream (0)>>> (cuda_cur_buffer (0), DIM);

  ret = hipStreamSynchronize (hip_stream (0));
  check (ret, "hipStreamSynchronize");

  monitoring_end_tile (clock, 0, 0, DIM, DIM, easypap_gpu_lane (0));

  return 0;
}
